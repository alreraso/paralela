#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>cc

#define ITERACIONES  2e09  
#define NUMERO_BLOQUES 40 
#define NUMERO_HILOS 1
int id_hilo;


struct transform4kto480_struct{
    Mat image;
    Mat result;
};


__global__ void* transform4kto480(void* arg,int iteraciones, int hilos, int bloques){
    int i;
	int index = blockIdx.x*blockDim.x+threadIdx.x;
	for (i=index; i< numero_iteraciones; i+=numero_hilos*numero_bloques) {

        struct transform4kto480_struct *arg_struct = (structtransform4kto480_struct*) arg;
        
        if(arg_struct->image.empty()) {
            cout << "Error: the arg_struct->image has been incorrectly loaded." << endl;
        }

        Mat temp(arg_struct->image.rows + 2, arg_struct->image.cols + 2, CV_8UC3, Scalar(255,255, 255));
        
        Mat copy( arg_struct->image.rows*2/9, arg_struct->image.cols/6, CV_8UC3, Scalar(255,255, 255));

        Vec3b cpixel;
        cpixel[0] = (uchar) 0;
        cpixel[1] = (uchar) 0;
        cpixel[2] = (uchar) 0;

        temp.at<Vec3b>(0, 0) = cpixel;
        temp.at<Vec3b>(temp.rows - 1, 0) = cpixel;
        temp.at<Vec3b>(0, temp.cols - 1) = cpixel;
        temp.at<Vec3b>(temp.rows - 1, temp.cols - 1) = cpixel;


        for(int i = 0; i < arg_struct->image.rows ; i++) {
            for(int j = 0; j < arg_struct->image.cols; j++) {
                cpixel = arg_struct->image.at<Vec3b>(i, j);
                temp.at<Vec3b>(i+1, j+1) = cpixel;
            }
        }

        for(int i = 0; i < arg_struct->image.rows; i++){
            cpixel = arg_struct->image.at<Vec3b>(i, 0);
            temp.at<Vec3b>(i+1, 0) = cpixel;
        }

        for(int i = 0; i < arg_struct->image.rows; i++){
            cpixel = arg_struct->image.at<Vec3b>(i, arg_struct->image.cols - 1);
            temp.at<Vec3b>(i+1, temp.cols - 1) = cpixel;
        }

        for(int i = 0; i < arg_struct->image.cols; i++){
            cpixel = arg_struct->image.at<Vec3b>(0, i);
            temp.at<Vec3b>(0, i + 1) = cpixel;
        }

        for(int i = 0; i < arg_struct->image.cols; i++){
            cpixel = arg_struct->image.at<Vec3b>(arg_struct->image.rows - 1, i);
            temp.at<Vec3b>(temp.rows - 1, i + 1) = cpixel;
        }

        for(int i = 0; i < arg_struct->image.rows; i++){
            for(int j = 0; j < arg_struct->image.cols; j++){
                Vec3b mpixel = temp.at<Vec3b>(i+1, j+1);
                Vec3b upixel = temp.at<Vec3b>(i, j+1);
                Vec3b dpixel = temp.at<Vec3b>(i+2, j+1);
                Vec3b lpixel = temp.at<Vec3b>(i+1, j);
                Vec3b rpixel = temp.at<Vec3b>(i+1, j+2);

                uchar a = (mpixel[0] + upixel[0] + dpixel[0] + lpixel[0] + rpixel[0])/5;
                uchar b = (mpixel[1] + upixel[1] + dpixel[1] + lpixel[1] + rpixel[1])/5;
                uchar c = (mpixel[2] + upixel[2] + dpixel[2] + lpixel[2] + rpixel[2])/5;

                Vec3b ppixel;
                ppixel[0] = a;
                ppixel[1] = b;
                ppixel[2] = c;

                if((i+j)%2 == 0){
                    if(i%2 == 0)
                        copy.at<Vec3b>((i*2)/9,j/6) = ppixel;
                    else
                        copy.at<Vec3b>(((i*2)/9)+1, j/6+1) = ppixel;
                }
            }
        }

        arg_struct->result = copy;
    }
}



int main(void) {
    struct timeval tval_before, tval_after, tval_result;
    gettimeofday(&tval_before, NULL); 

	dim3 dimGrid(NUMERO_BLOQUES,1,1); 
	dim3 dimBlock(NUMERO_HILOS,1,1);
    struct transform4kto480_struct *arg_struct;
    struct transform4kto480_struct host = (void *)malloc(size);
	size_t size = NUMERO_BLOQUES*NUMERO_HILOS*sizeof(double);

	hipMalloc((void **) &arg_struct, size);
	hipMemset(arg_struct, 0, size);

    transform4kto480 <<<dimGrid, dimBlock>>> (arg_struct, ITERACIONES, NUMERO_HILOS, NUMERO_BLOQUES);
	
	hipMemcpy(host, arg_struct, size, hipMemcpyDeviceToHost);

    gettimeofday(&tval_after, NULL);
    timersub(&tval_after, &tval_before, &tval_result);

    printf("Tiempo transcurrido: %ld.%06ld segundos\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);

    free(host);
	hipFree(arg_struct);

	return 0;
}