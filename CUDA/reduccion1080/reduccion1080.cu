#include "hip/hip_runtime.h"
//standard libraries
#include <iostream>
#include <stdio.h>
#include <sys/time.h>
#include <math.h>
#include <cstdio>
#include <cmath>
#include <string>
#include <ctime>
#include <stdlib.h>
#include <unistd.h>
#include <fstream>
//opencv libraries
//#include <opencv2/core/core.hpp>
//#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
//CUDA libraries
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "opencv2/gpu/gpu.hpp"
//#include "opencv2/gpu.hpp"
#include <>

using namespace std;
using namespace cv;

__global__ void transform1080to480(Mat *image, string *result_image, int n);

int main(int argc, char** argv) {

    if (argc < 5) {
        // Tell the user how to run the program
        cerr << "Uso:" << argv[0] << " Imagen-Entrada Imagen-Salida #Hilos #Bloques(Ejemplo:./reduccion1080 1080.jpg result.jpg 256 8)"<< endl;
        /* "Usage messages" are a conventional way of telling the user
         * how to run a program if they enter the command incorrectly.
         */
        return 1;
    }
    
    //Size of vectors
    long n = 100000000;

    //Host input
    //Mat *h_image = imread(argv[1], IMREAD_COLOR);
    Mat *h_image;
    //Host output
    //string *h_result_image = argv[2];
    string *h_result_image;
    //Device input
    //Mat *d_image = imread(argv[1], IMREAD_COLOR);
    Mat *d_image;
    //Device output
    //string *d_result_image = argv[2];
    string *d_result_image

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(Mat);

    struct timeval tval_before, tval_after, tval_result;

    gettimeofday(&tval_before, NULL);

    // Allocate memory on host
    h_image = (Mat*)malloc(bytes);
    h_result_image = (Mat*)malloc(bytes);

    // Allocate memory on GPU
    hipMalloc(&d_image, bytes);
    hipMalloc(&d_result_image, bytes);

    //Initialize on host
    Mat h_image = imread(argv[1], IMREAD_COLOR);

    // Copy host to device
    hipMemcpy( d_image, h_image, bytes, hipMemcpyHostToDevice);

    int THREADS, BLOCKS;

    // Number of threads in each thread block
    int THREADS = atoi(argv[3]);
     // Number of thread blocks in grid
    int BLOCKS = atoi(argv[4]);

    // Execute the kernel
    transform1080to480<<<gridSize, blockSize>>>(d_image, d_result_image, n);
 
    // Copy array back to host
    hipMemcpy( h_result_image, d_result_image, bytes, hipMemcpyDeviceToHost );

    // Release device memory
    hipFree(d_image);
    hipFree(d_result_image);
 
    // Release host memory
    free(h_image);
    free(h_result_image);

    gettimeofday(&tval_after, NULL);

    timersub(&tval_after,&tval_before,&tval_result);

    FILE * pFile;
    pFile = fopen("/../../resultados.txt", "a");
    fprintf(pFile, "Time elapsed transforming a 1080p image to 480p using CUDA with %d threads and %d blocks: %ld.%06lds\n", THREADS, BLOCKS, (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);    
    fclose(pFile);    
    return 0;
}


__global__ void transform1080to480(Mat *image, string *result_image, int n){

    if(image.empty()) {
        cout << "Error: the image has been incorrectly loaded." << endl;
    }

    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    Mat temp(image.rows + 2, image.cols + 2, CV_8UC3, Scalar(255,255, 255));

    Mat copy( (image.rows*4)/9, image.cols/3, CV_8UC3, Scalar(255,255, 255));

    if (id < n){    

    Vec3b cpixel;
    cpixel[0] = (uchar) 0;
    cpixel[1] = (uchar) 0;
    cpixel[2] = (uchar) 0;

    temp.at<Vec3b>(0, 0) = cpixel;
    temp.at<Vec3b>(temp.rows - 1, 0) = cpixel;
    temp.at<Vec3b>(0, temp.cols - 1) = cpixel;
    temp.at<Vec3b>(temp.rows - 1, temp.cols - 1) = cpixel;


    for(int i = 0; i < image.rows ; i++) {
        for(int j = 0; j < image.cols; j++) {
            cpixel = image.at<Vec3b>(i, j);
            temp.at<Vec3b>(i+1, j+1) = cpixel;
        }
    }

    for(int i = 0; i < image.rows; i++){
        cpixel = image.at<Vec3b>(i, 0);
        temp.at<Vec3b>(i+1, 0) = cpixel;
    }

    for(int i = 0; i < image.rows; i++){
        cpixel = image.at<Vec3b>(i, image.cols - 1);
        temp.at<Vec3b>(i+1, temp.cols - 1) = cpixel;
    }

    for(int i = 0; i < image.cols; i++){
        cpixel = image.at<Vec3b>(0, i);
        temp.at<Vec3b>(0, i + 1) = cpixel;
    }

    for(int i = 0; i < image.cols; i++){
        cpixel = image.at<Vec3b>(image.rows - 1, i);
        temp.at<Vec3b>(temp.rows - 1, i + 1) = cpixel;
    }

    for(int i = 0; i < image.rows; i++){
        for(int j = 0; j < image.cols; j++){
            Vec3b mpixel = temp.at<Vec3b>(i+1, j+1);
            Vec3b upixel = temp.at<Vec3b>(i, j+1);
            Vec3b dpixel = temp.at<Vec3b>(i+2, j+1);
            Vec3b lpixel = temp.at<Vec3b>(i+1, j);
            Vec3b rpixel = temp.at<Vec3b>(i+1, j+2);

            uchar a = (mpixel[0] + upixel[0] + dpixel[0] + lpixel[0] + rpixel[0])/5;
            uchar b = (mpixel[1] + upixel[1] + dpixel[1] + lpixel[1] + rpixel[1])/5;
            uchar c = (mpixel[2] + upixel[2] + dpixel[2] + lpixel[2] + rpixel[2])/5;

            Vec3b ppixel;
            ppixel[0] = a;
            ppixel[1] = b;
            ppixel[2] = c;

            if((i+j)%2 == 0){
                if(i%2 == 0)
                    copy.at<Vec3b>((i*4)/9,j/3) = ppixel;
                else
                    copy.at<Vec3b>(((i*4)/9)+1, j/3+1) = ppixel;
            }
        }
    }    
        
        imwrite(result_image, copy);
/*  Then we create a window to display our image
    namedWindow("My first OpenCV window");

    // Finally, we display our image and ask the program to wait for a key to be pressed
    imshow("My first OpenCV window", copy);
    waitKey(0);
*/}
}