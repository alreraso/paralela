#include "hip/hip_runtime.h"
#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <iostream>
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace std;
using namespace cv;


__global__ void transform4kto480(Mat *image, int *ID, string *result_image);

int main(int argc, char** argv) {

    if (argc < 5) {
        // Tell the user how to run the program
        cerr << "Uso:" << argv[0] << " Imagen-Entrada Imagen-Salida #Hilos #Bloques(Ejemplo:./reduccion4k 4k.jpg result.jpg 1024 16)"<< endl;
        /* "Usage messages" are a conventional way of telling the user
         * how to run a program if they enter the command incorrectly.
         */
        return 1;
    }

    Mat *image = imread(argv[1], IMREAD_COLOR);

    Mat *copy;

    string result_image = argv[2];

    int THREADS = atoi(argv[3]);

    int N = atoi(argv[4]); //BLOQUES

    Mat *d_image;

    Mat *d_copy;

    Mat size = N * sizeof(Mat);

    hipMalloc((void **)&d_image, size);

    image = (Mat *)malloc(size); random_ints(image, N);
    
    copy = (Mat *)malloc(size);

    hipMemcpy(d_image, image, size, hipMemcpyHostToDevice);

    struct timeval tval_before, tval_after, tval_result;

    gettimeofday(&tval_before, NULL);    
    
    transform4kto480<<<N,1>>>(d_image);
    // Copy result back to host
    hipMemcpy(copy, d_copy, size, hipMemcpyDeviceToHost);
    // Cleanup
    free(image); 
    free(copy);
    hipFree(d_image); 
    hipFree(d_copy);
    
    gettimeofday(&tval_after, NULL);

    timersub(&tval_after,&tval_before,&tval_result);

    FILE * pFile;
    pFile = fopen("resultados.txt", "a");
    fprintf(pFile, "Time elapsed transforming a 4k image to 480p using CUDA with %d threads and %d blocks: %ld.%06lds\n", THREADS, BLOCKS, (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);    
    fclose(pFile);    
    return 0;
}

__global__ void transform4kto480(Mat *image, int *ID, string *result_image){

    int index = threadIdx.x + blockIdx.x * blockDim.x;

    image[index]

    if(image[blockIdx.x].empty()) {
        cout << "Error: the image has been incorrectly loaded." << endl;
    }

    Mat temp[blockIdx.x](image[blockIdx.x].rows + 2, image[blockIdx.x].cols + 2, CV_8UC3, Scalar(255,255, 255));

    Mat copy[blockIdx.x]( (image[blockIdx.x].rows*4)/9, image[blockIdx.x].cols/3, CV_8UC3, Scalar(255,255, 255));
    

    Vec3b cpixel;
    cpixel[0] = (uchar) 0;
    cpixel[1] = (uchar) 0;
    cpixel[2] = (uchar) 0;

    temp[blockIdx.x].at<Vec3b>(0, 0) = cpixel;
    temp[blockIdx.x].at<Vec3b>(temp[blockIdx.x].rows - 1, 0) = cpixel;
    temp[blockIdx.x].at<Vec3b>(0, temp[blockIdx.x].cols - 1) = cpixel;
    temp[blockIdx.x].at<Vec3b>(temp[blockIdx.x].rows - 1, temp[blockIdx.x].cols - 1) = cpixel;


    for(int i = 0; i < image[blockIdx.x].rows ; i++) {
        for(int j = 0; j < image[blockIdx.x].cols; j++) {
            cpixel = image[blockIdx.x].at<Vec3b>(i, j);
            temp[blockIdx.x].at<Vec3b>(i+1, j+1) = cpixel;
        }
    }

    for(int i = 0; i < image[blockIdx.x].rows; i++){
        cpixel = image[blockIdx.x].at<Vec3b>(i, 0);
        temp[blockIdx.x].at<Vec3b>(i+1, 0) = cpixel;
    }

    for(int i = 0; i < image[blockIdx.x].rows; i++){
        cpixel = image[blockIdx.x].at<Vec3b>(i, image[blockIdx.x].cols - 1);
        temp[blockIdx.x].at<Vec3b>(i+1, temp[blockIdx.x].cols - 1) = cpixel;
    }

    for(int i = 0; i < image[blockIdx.x].cols; i++){
        cpixel = image[blockIdx.x].at<Vec3b>(0, i);
        temp[blockIdx.x].at<Vec3b>(0, i + 1) = cpixel;
    }

    for(int i = 0; i < image[blockIdx.x].cols; i++){
        cpixel = image[blockIdx.x].at<Vec3b>(image[blockIdx.x].rows - 1, i);
        temp[blockIdx.x].at<Vec3b>(temp[blockIdx.x].rows - 1, i + 1) = cpixel;
    }

    for(int i = 0; i < image[blockIdx.x].rows; i++){
        for(int j = 0; j < image[blockIdx.x].cols; j++){
            Vec3b mpixel = temp[blockIdx.x].at<Vec3b>(i+1, j+1);
            Vec3b upixel = temp[blockIdx.x].at<Vec3b>(i, j+1);
            Vec3b dpixel = temp[blockIdx.x].at<Vec3b>(i+2, j+1);
            Vec3b lpixel = temp[blockIdx.x].at<Vec3b>(i+1, j);
            Vec3b rpixel = temp[blockIdx.x].at<Vec3b>(i+1, j+2);

            uchar a = (mpixel[0] + upixel[0] + dpixel[0] + lpixel[0] + rpixel[0])/5;
            uchar b = (mpixel[1] + upixel[1] + dpixel[1] + lpixel[1] + rpixel[1])/5;
            uchar c = (mpixel[2] + upixel[2] + dpixel[2] + lpixel[2] + rpixel[2])/5;

            Vec3b ppixel;
            ppixel[0] = a;
            ppixel[1] = b;
            ppixel[2] = c;

            if((i+j)%2 == 0){
                if(i%2 == 0)
                    copy[blockIdx.x].at<Vec3b>((i*4)/9,j/3) = ppixel;
                else
                    copy[blockIdx.x].at<Vec3b>(((i*4)/9)+1, j/3+1) = ppixel;
            }
        }
    }    
        
        imwrite(*result_image, copy[blockIdx.x]);
/*  Then we create a window to display our image
    namedWindow("My first OpenCV window");

    // Finally, we display our image and ask the program to wait for a key to be pressed
    imshow("My first OpenCV window", *copy);
    waitKey(0);
*/
}