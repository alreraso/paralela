#include "hip/hip_runtime.h"
//standard libraries
#include <iostream>
#include <stdio.h>
#include <sys/time.h>
//opencv libraries
#include <opencv2/opencv.hpp>
#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
//CUDA libraries
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "opencv2/core/cuda.hpp"
#include "opencv2/core/cuda_types.hpp"
#include "opencv2/core/cuda_stream_accessor.hpp"
#include <>


using namespace std;
using namespace cv;

__global__ void transform1080to480(Mat *image, string *result_image, int n);

int main(int argc, char** argv) {

    if (argc < 5) {
        // Tell the user how to run the program
        cerr << "Uso:" << argv[0] << " Imagen-Entrada Imagen-Salida #Hilos #Bloques(Ejemplo:./reduccion720 720.jpg result.jpg 256 8)"<< endl;
        /* "Usage messages" are a conventional way of telling the user
         * how to run a program if they enter the command incorrectly.
         */
        return 1;
    }
    
    if (atoi(argv[4]) <= 0)
    {
        printf("Por favor use un numero positivo de bloques\n");
        return 1;
    }

    if (atoi(argv[3]) <= 0)
    {
        printf("Por favor use un numero positivo de hilos\n");
        return 1;
    }

    //Size of vectors
    long n = 100000;

    // Size, in bytes, of each vector
    size_t mat_size = n*sizeof(Mat);
    size_t string_size = n*sizeof(string);

    struct timeval tval_before, tval_after, tval_result;

    gettimeofday(&tval_before, NULL);

    // Allocate memory on host
    h_image = (Mat*)malloc(mat_size);
    h_result_image = (string*)malloc(string_size);

    // Allocate memory on GPU
    hipMalloc(&d_image, mat_size);
    hipMalloc(&d_result_image, string_size);

    //Initialize on host
    h_image = imread(argv[1], IMREAD_COLOR);

    // Copy host to device
    hipMemcpy( d_image, h_image, mat_size, hipMemcpyHostToDevice);

    int THREADS, BLOCKS;

    //Host input
    Mat *h_image = imread(argv[1], IMREAD_COLOR);
    
    //Host output
    string *h_result_image = argv[2];
    
    //Device input
    Mat *d_image = imread(argv[1], IMREAD_COLOR);
    
    //Device output
    string *d_result_image = argv[2];

    // Number of threads in each thread block
    THREADS = atoi(argv[3]);
     // Number of thread blocks in grid
    BLOCKS = atoi(argv[4]);

    // Execute the kernel
    transform1080to480<<<BLOCKS, THREADS>>>(d_image, d_result_image, n);
 
    // Copy array back to host
    hipMemcpy( h_result_image, d_result_image, string_size, hipMemcpyDeviceToHost );

    // Release device memory
    hipFree(d_image);
    hipFree(d_result_image);
 
    // Release host memory
    free(h_image);
    free(h_result_image);

    gettimeofday(&tval_after, NULL);

    timersub(&tval_after,&tval_before,&tval_result);

    FILE * pFile;
    pFile = fopen("/../../resultados.txt", "a");
    fprintf(pFile, "Time elapsed transforming a 1080p image to 480p using CUDA with %d threads and %d blocks: %ld.%06lds\n", THREADS, BLOCKS, (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);    
    fclose(pFile);    
    return 0;
}


__global__ void transform1080to480(Mat *image, string *result_image, int n){

    if(image.empty()) {
        cout << "Error: the image has been incorrectly loaded." << endl;
    }

    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    Mat temp(image.rows + 2, image.cols + 2, CV_8UC3, Scalar(255,255, 255));

    Mat copy( (image.rows*2)/3, image.cols/2, CV_8UC3, Scalar(255,255, 255));    

    Vec3b cpixel;
    cpixel[0] = (uchar) 0;
    cpixel[1] = (uchar) 0;
    cpixel[2] = (uchar) 0;

    temp.at<Vec3b>(0, 0) = cpixel;
    temp.at<Vec3b>(temp.rows - 1, 0) = cpixel;
    temp.at<Vec3b>(0, temp.cols - 1) = cpixel;
    temp.at<Vec3b>(temp.rows - 1, temp.cols - 1) = cpixel;


    for(int i = 0; i < image.rows ; i++) {
        for(int j = 0; j < image.cols; j++) {
            cpixel = image.at<Vec3b>(i, j);
            temp.at<Vec3b>(i+1, j+1) = cpixel;
        }
    }

    for(int i = 0; i < image.rows; i++){
        cpixel = image.at<Vec3b>(i, 0);
        temp.at<Vec3b>(i+1, 0) = cpixel;
    }

    for(int i = 0; i < image.rows; i++){
        cpixel = image.at<Vec3b>(i, image.cols - 1);
        temp.at<Vec3b>(i+1, temp.cols - 1) = cpixel;
    }

    for(int i = 0; i < image.cols; i++){
        cpixel = image.at<Vec3b>(0, i);
        temp.at<Vec3b>(0, i + 1) = cpixel;
    }

    for(int i = 0; i < image.cols; i++){
        cpixel = image.at<Vec3b>(image.rows - 1, i);
        temp.at<Vec3b>(temp.rows - 1, i + 1) = cpixel;
    }

    for(int i = 0; i < image.rows; i++){
        for(int j = 0; j < image.cols; j++){
            Vec3b mpixel = temp.at<Vec3b>(i+1, j+1);
            Vec3b upixel = temp.at<Vec3b>(i, j+1);
            Vec3b dpixel = temp.at<Vec3b>(i+2, j+1);
            Vec3b lpixel = temp.at<Vec3b>(i+1, j);
            Vec3b rpixel = temp.at<Vec3b>(i+1, j+2);

            uchar a = (mpixel[0] + upixel[0] + dpixel[0] + lpixel[0] + rpixel[0])/5;
            uchar b = (mpixel[1] + upixel[1] + dpixel[1] + lpixel[1] + rpixel[1])/5;
            uchar c = (mpixel[2] + upixel[2] + dpixel[2] + lpixel[2] + rpixel[2])/5;

            Vec3b ppixel;
            ppixel[0] = a;
            ppixel[1] = b;
            ppixel[2] = c;

            if((i+j)%2 == 0){
                if(i%2 == 0)
                    copy.at<Vec3b>((i*2)/3,j/2) = ppixel;
                else
                    copy.at<Vec3b>(((i*2)/3)+1, j/2+1) = ppixel;
            }
        }
    }    
        //Write resized image
        imwrite(result_image, copy);
}