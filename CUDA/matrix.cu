#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N (2048*2048);
#define THREADS_PER_BLOCK 512;

__global__ void add(int *a, int *b, int *c, int n);

int main (void) {
    int *a, *b, *c //host copies of a, b and c
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    int size = N * sizeof(int);
    // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    // Alloc space for host copies of a, b, c and setup input values
    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size);
    
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
    //add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
    add<<<(N + M-1) / M,M>>>(d_a, d_b, d_c, N);
    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}

__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
     c[index] = a[index] + b[index];
}
    