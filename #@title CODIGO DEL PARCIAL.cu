#include "hip/hip_runtime.h"
#@title CODIGO DEL PARCIAL
%%cu
/****** codigo para calcular PI*******/
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define ITERACIONES  2e09  
#define NUMERO_BLOQUES 8
#define NUMERO_HILOS 1032
int id_hilo;
double pi = 0;

__global__ void calcular_pi(double *suma, int numero_iteraciones, double paso, int numero_hilos, int numero_bloques) {
	int i;
	double x;
	int index = blockIdx.x*blockDim.x+threadIdx.x;
	for (i=index; i< numero_iteraciones; i+=numero_hilos*numero_bloques) {
		x = (i+0.5)*paso;
		suma[index] += 4.0/(1.0+x*x);
	}
}

int main(void) {
    
    struct timeval tval_before, tval_after, tval_result;
    gettimeofday(&tval_before, NULL); 

	dim3 dimGrid(NUMERO_BLOQUES,1,1); 
	dim3 dimBlock(NUMERO_HILOS,1,1);
	double *sumaHost, *sumaDispositivo;
	double paso = 1.0/ITERACIONES;
	size_t size = NUMERO_BLOQUES*NUMERO_HILOS*sizeof(double);
    
    sumaHost = (double *)malloc(size); 
	hipMalloc((void **) &sumaDispositivo, size);
	hipMemset(sumaDispositivo, 0, size);
    
    calcular_pi <<<dimGrid, dimBlock>>> (sumaDispositivo, ITERACIONES, paso, NUMERO_HILOS, NUMERO_BLOQUES);
	
	hipMemcpy(sumaHost, sumaDispositivo, size, hipMemcpyDeviceToHost);
	for(id_hilo=0; id_hilo<NUMERO_HILOS*NUMERO_BLOQUES; id_hilo++)
		pi += sumaHost[id_hilo];
	pi *= paso;

    gettimeofday(&tval_after, NULL);
    timersub(&tval_after, &tval_before, &tval_result);

    printf("Tiempo transcurrido: %ld.%06ld segundos\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);

	printf("PI = %lf\n",pi);

	free(sumaHost); 
	hipFree(sumaDispositivo);

	return 0;
}