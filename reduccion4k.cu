#include "hip/hip_runtime.h"
#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <iostream>
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

int id_hilo;

using namespace std;
using namespace cv;

__global__ void transform4kto480(Mat image, string result_image);

int main(int argc, char** argv) {

    if (argc < 5) {
        // Tell the user how to run the program
        cerr << "Uso:" << argv[0] << " Imagen-Entrada Imagen-Salida #Hilos #Bloques(Ejemplo:./reduccion4k 4k.jpg result.jpg 1024 16)"<< endl;
        /* "Usage messages" are a conventional way of telling the user
         * how to run a program if they enter the command incorrectly.
         */
        return 1;
    }

    Mat image = imread(argv[1], IMREAD_COLOR);

    string result_image = argv[2];

    int THREADS = atoi(argv[3]);

    int BLOCKS = atoi(argv[4]);

    struct timeval tval_before, tval_after, tval_result;

    gettimeofday(&tval_before, NULL);
    
    dim3 dimGrid(BLOCKS,1,1); 
	dim3 dimBlock(THREADS,1,1);
	double *sumaHost, *sumaDispositivo;
    size_t size = BLOCKS*THREADS*sizeof(double);
    
    sumaHost = (double *)malloc(size); 
	hipMalloc((void **) &sumaDispositivo, size);
    hipMemset(sumaDispositivo, 0, size);
    
    transform4kto480 <<<dimGrid, dimBlock>>> (sumaDispositivo, BLOCKS, THREADS);
    hipMemcpy(sumaHost, sumaDispositivo, size, hipMemcpyDeviceToHost);
    for(id_hilo=0; id_hilo<THREADS*BLOCKS; id_hilo++)
		pi += sumaHost[id_hilo];
	pi *= paso;
    
    gettimeofday(&tval_after, NULL);

    timersub(&tval_after,&tval_before,&tval_result);

    FILE * pFile;
    pFile = fopen("resultados.txt", "a");
    fprintf(pFile, "Time elapsed transforming a 4k image to 480p using CUDA with %d threads and %d blocks: %ld.%06lds\n", THREADS, BLOCKS, (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);    
    fclose(pFile);    
    return 0;
}


__global__ void transform4kto480(Mat image, string result_image){
    int i;
    double x;
    int index = blockIdx.x*blockDim.x+threadIdx.x;
	for (i=index; i< numero_iteraciones; i+=THREADS*BLOCKS) {
		x = (i+0.5)*paso;
		suma[index] += 4.0/(1.0+x*x);
	}

    if(image.empty()) {
        cout << "Error: the image has been incorrectly loaded." << endl;
    }

    Mat temp(image.rows + 2, image.cols + 2, CV_8UC3, Scalar(255,255, 255));
    
    //tamaño de 4k a 480
    Mat copy( image.rows*2/9, image.cols/6, CV_8UC3, Scalar(255,255, 255));

    Vec3b cpixel;
    cpixel[0] = (uchar) 0;
    cpixel[1] = (uchar) 0;
    cpixel[2] = (uchar) 0;

    temp.at<Vec3b>(0, 0) = cpixel;
    temp.at<Vec3b>(temp.rows - 1, 0) = cpixel;
    temp.at<Vec3b>(0, temp.cols - 1) = cpixel;
    temp.at<Vec3b>(temp.rows - 1, temp.cols - 1) = cpixel;    

        for(int i = 0; i < image.rows ; i++) {
            for(int j = 0; j < image.cols; j++) {
                cpixel = image.at<Vec3b>(i, j);
                temp.at<Vec3b>(i+1, j+1) = cpixel;
            }
        }

        for(int i = 0; i < image.rows; i++){
            cpixel = image.at<Vec3b>(i, 0);
            temp.at<Vec3b>(i+1, 0) = cpixel;
        }

        for(int i = 0; i < image.rows; i++){
            cpixel = image.at<Vec3b>(i, image.cols - 1);
            temp.at<Vec3b>(i+1, temp.cols - 1) = cpixel;
        }

        for(int i = 0; i < image.cols; i++){
            cpixel = image.at<Vec3b>(0, i);
            temp.at<Vec3b>(0, i + 1) = cpixel;
        }

        for(int i = 0; i < image.cols; i++){
            cpixel = image.at<Vec3b>(image.rows - 1, i);
            temp.at<Vec3b>(temp.rows - 1, i + 1) = cpixel;
        }

        for(int i = 0; i < image.rows; i++){
            for(int j = 0; j < image.cols; j++){
                Vec3b mpixel = temp.at<Vec3b>(i+1, j+1);
                Vec3b upixel = temp.at<Vec3b>(i, j+1);
                Vec3b dpixel = temp.at<Vec3b>(i+2, j+1);
                Vec3b lpixel = temp.at<Vec3b>(i+1, j);
                Vec3b rpixel = temp.at<Vec3b>(i+1, j+2);

                uchar a = (mpixel[0] + upixel[0] + dpixel[0] + lpixel[0] + rpixel[0])/5;
                uchar b = (mpixel[1] + upixel[1] + dpixel[1] + lpixel[1] + rpixel[1])/5;
                uchar c = (mpixel[2] + upixel[2] + dpixel[2] + lpixel[2] + rpixel[2])/5;

                Vec3b ppixel;
                ppixel[0] = a;
                ppixel[1] = b;
                ppixel[2] = c;

                if((i+j)%2 == 0){
                    if(i%2 == 0)
                        copy.at<Vec3b>((i*2)/9,j/6) = ppixel;
                    else
                        copy.at<Vec3b>(((i*2)/9)+1, j/6+1) = ppixel;
                }
            }
        }    
        
        imwrite(result_image, copy);
}